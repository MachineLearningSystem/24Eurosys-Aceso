#include "super_scaler.h"


void initialization(int &myRank, int &nRanks, int &localRank)
{
    //initializing MPI
    MPICHECK(MPI_Init(NULL, NULL));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

    global_cfg.parse_excution_plan("configure/configure.cfg");

    //calculating localRank which is used in selecting a GPU
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));

    for (int p = 0; p < nRanks; p++)
    {
        if (p == myRank)
        {
            break;
        }
        if (hostHashs[p] == hostHashs[myRank])
        {
            localRank++;
        }
    }
}

void finalization()
{
    //finalizing MPI
    MPICHECK(MPI_Finalize());
}


void MPI_scaler_all_reduce_host(float *gradients, int size, int myRank, int nRanks, int localRank)
{
    MPICHECK(MPI_Allreduce(MPI_IN_PLACE, (void *)gradients,
        (int) size,
        MPI_FLOAT,
        MPI_SUM,
        MPI_COMM_WORLD));
    
    for (int i = 0; i < size; i++)
    {
        gradients[i] /= nRanks;
    }
    //call back
    //(*callback)();
}

void MPI_usr_scaler_all_reduce_host(float *gradients, int size, int myRank, int nRanks, int localRank,
                                    plan plan, void* output_ptr)
{

    //auto plan = global_cfg.cfg_table["allreduce.classifier.6.bias"];

    MPI_Status recv_status;
    MPI_Request recv_req;
    //void* output_ptr = malloc(size*sizeof(float));
    float* output = (float*)output_ptr;

    for(auto op_ :plan.operation)
    {
        if(op_.operation_type == "send_receive")
        {
            if(op_.average)
            {
                float* segment_send = (float*)gradients + op_.send_address[myRank];
                float* segment_receive = (float*)gradients + op_.receive_address[myRank];
                float* segment_receive2 = output + op_.receive_address[myRank];
                MPI_Irecv(segment_receive2, op_.receive_length[myRank],
                        MPI_FLOAT, 
                        op_.receive_target[myRank], 
                        0, MPI_COMM_WORLD, &recv_req);
                MPI_Send(segment_send, op_.send_length[myRank],
                        MPI_FLOAT, 
                        op_.send_target[myRank], 
                        0, MPI_COMM_WORLD);

                MPI_Wait(&recv_req, &recv_status);

                for(int i = 0 ; i < op_.receive_length[myRank]; i++)
                    segment_receive[i] += segment_receive2[i];
            }
            else
            {
                float* segment_send = (float*)gradients + op_.send_address[myRank];
                float* segment_receive = (float*)gradients + op_.receive_address[myRank];
                MPI_Sendrecv(segment_send, op_.send_length[myRank],
                             MPI_FLOAT,
                             op_.send_target[myRank], 0,
                             segment_receive, op_.receive_length[myRank],
                             MPI_FLOAT,
                             op_.receive_target[myRank], 
                             0, MPI_COMM_WORLD, &recv_status);
            }
        }
        else if(op_.operation_type == "send")
          {
            if(op_.send_target[myRank] == -1)
              continue;
            else
            {
              float* segment_send = (float*)gradients + op_.send_address[myRank];
              MPI_Send(segment_send, op_.send_length[myRank],
                       MPI_FLOAT, 
                       op_.send_target[myRank], 0, MPI_COMM_WORLD);
            }
          }
          else if(op_.operation_type == "receive")
          {
            if(op_.receive_target[myRank] == -1)
              continue;
            else
            {
              if(op_.average){
                float* segment_receive = (float*)gradients + op_.receive_address[myRank];
                float* segment_receive2 = output + op_.receive_address[myRank];
                MPI_Irecv(segment_receive2, op_.receive_length[myRank],
                          MPI_FLOAT, 
                          op_.receive_target[myRank], 0, MPI_COMM_WORLD, &recv_req);
                MPI_Wait(&recv_req, &recv_status);
                for(int i = 0 ; i < op_.receive_length[myRank]; i++)
                    segment_receive[i] += segment_receive2[i];
              }
              else
              {
                float* segment_receive = (float*)gradients + op_.receive_address[myRank];
                MPI_Recv(segment_receive, op_.receive_length[myRank],
                         MPI_FLOAT, 
                         op_.receive_target[myRank], 0, MPI_COMM_WORLD, &recv_status);
              }
              
            }
          }
    }
    //delete[] output;
    
    for (int i = 0; i < size; i++)
    {
        gradients[i] /= nRanks;
    }
    //call back
    //(*callback)();
}

void nccl_super_scaler_all_reduce_host(float *gradients, int size, int myRank, int nRanks, int localRank,
                                  float **sendbuff, float **recvbuff, ncclComm_t* comms, hipStream_t *s)
{
    //each process use 1 GPU
    int nDev = 1;

    for (int i = 0; i < nDev; ++i)
    {
        CUDACHECK(hipMemcpy(sendbuff[i], gradients, size * sizeof(float), hipMemcpyHostToDevice));
    }
    //calling NCCL communication API. Group API is required when using multiple devices per thread/process
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; i++)
    {
        NCCLCHECK(ncclAllReduce((const void *)sendbuff[i], (void *)recvbuff[i], size, ncclFloat, ncclSum, comms[i], s[i]));
    }
    NCCLCHECK(ncclGroupEnd());

    //synchronizing on CUDA stream to complete NCCL communication
    for (int i = 0; i < nDev; i++)
    {
        CUDACHECK(hipStreamSynchronize(s[i]));
    }

    //get gradients after allreduce
    for (int i = 0; i < nDev; i++)
    {
        //CUDACHECK(hipSetDevice(localRank * nDev + i));
        gradients_Average(recvbuff[i], size, nRanks);
        CUDACHECK(hipMemcpy(gradients, recvbuff[i], sizeof(float) * size, hipMemcpyDeviceToHost));
    }

    //(*callback)();
}

void nccl_super_scaler_all_reduce_device(float *gradients, int size, int myRank, int nRanks, int localRank,
                                    ncclComm_t* comms, hipStream_t *s)
{
    //each process use 1 GPU
    int nDev = 1;
    //calling NCCL communication API. Group API is required when using multiple devices per thread/process
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < nDev; i++)
    {
        NCCLCHECK(ncclAllReduce((const void *)gradients, (void *)gradients, size, ncclFloat, ncclSum, comms[i], s[i]));
    }
    NCCLCHECK(ncclGroupEnd());
    //synchronizing on CUDA stream to complete NCCL communication
    for (int i = 0; i < nDev; i++)
    {
        CUDACHECK(hipStreamSynchronize(s[i]));
    }

    //get gradients after allreduce
    for (int i = 0; i < nDev; i++)
    {
        //CUDACHECK(hipSetDevice(localRank * nDev + i));
        gradients_Average(gradients, size, nRanks);
        CUDACHECK(hipStreamSynchronize(s[i]));
    }
    
    //call back
    //(*callback)();
}
